
#include <hip/hip_runtime.h>
#include <stdio.h>

__global__ void print()
{
    printf("thread %d\n", threadIdx.x);
}

int main()
{
    print<<<1, 1>>>();
    hipDeviceSynchronize();
}