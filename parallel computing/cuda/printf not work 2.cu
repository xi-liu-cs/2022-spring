
#include <hip/hip_runtime.h>
#include <stdio.h>

__global__ void print(char ** dest)
{
    const char * temp = "123";
    int i;
    for(i = 0; i < 3; ++i)
        (*dest)[i] = temp[i];
    (*dest)[i] = '\0';
}

int main()
{
    char * str = (char *)malloc(4);
    print<<<1, 1>>>(&str);
    printf(str);
    hipDeviceSynchronize();
}
