
#include <hip/hip_runtime.h>
#include <stdio.h>

__global__ void print(char ** dstr)
{
    const char * temp = "123";
    int i;
    for(i = 0; i < 3; ++i)
        (*dstr)[i] = temp[i];
    (*dstr)[i] = '\0';
}

int main()
{
    char * hstr = (char *)malloc(4), * dstr;
    hipMalloc(&dstr, 4);
    print<<<1, 1>>>(&dstr);
    hipMemcpy(hstr, dstr, 4, hipMemcpyDeviceToHost);
    printf(hstr);
    hipDeviceSynchronize();
}
