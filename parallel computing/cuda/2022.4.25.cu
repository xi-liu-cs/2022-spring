#include <stdio.h>
#include <hip/hip_runtime.h>

__global__ void vec_add(float * x, float * y, float * z, int n)
{
    int i = blockIdx.x * blockDim.x + threadIdx.x;
    if(i < n)
        z[i] = x[i] + y[i];
}

void alloc_vec(float ** xp, float ** yp, float ** zp, int n)
{
    hipMallocManaged(xp, n * sizeof(float));
    hipMallocManaged(yp, n * sizeof(float));
    hipMallocManaged(zp, n * sizeof(float));
}

__global__ void init_vec(float * x, float * y)
{
    float a1[] = {1.0, 2.0, 3.0},
    a2[] = {1.0, 2.0, 3.0};
    int n = sizeof(a1) / sizeof(*a1);
    for(int i = 0; i < n; ++i)
    {
        x[i] = a1[i];
        y[i] = a2[i];
    }   
}

void print(float * a, int n)
{
    for(int i = 0; i < n; ++i)
        printf("%f ", a[i]);
    printf("\n");
}

int main()
{
    int blk_ct = 1 /* block_count */,
    th_per_blk = 1 /* thread_per_block */;
    int n = 3;
    float * x, * y, * z;
    alloc_vec(&x, &y, &z, n);
    init_vec<<<blk_ct, th_per_blk>>>(x, y);
    vec_add<<<blk_ct, th_per_blk>>>(x, y, z, n);
    print(z, n);   
    hipDeviceSynchronize();
}