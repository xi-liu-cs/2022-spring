#include <stdio.h>
#include <stdlib.h>
#include <hip/hip_runtime.h>

__global__ void add(int a, int b, int * dsum)
{
    *dsum = a + b;
}

int main()
{
    int a = 3, b = 2;
    int * hsum = (int *)malloc(sizeof(int)), * dsum;
    hipMalloc(&dsum, sizeof(int));
    add<<<1, 1>>>(a, b, dsum);
    hipMemcpy(hsum, dsum, sizeof(int), hipMemcpyDeviceToHost);
    printf("hsum = %d\n", *hsum);
    free(hsum);
    hipFree(dsum);
}
