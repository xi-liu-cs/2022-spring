#include <stdio.h>
#include <hip/hip_runtime.h>

__global__ void printing()
{
    __shared__ int x;
    x = 7;
    if(blockIdx.x > 0)
        printf("blk.x = %d\n", blockIdx.x);
    else
        printf("x = %d\n", x);
    __syncthreads();
    printf("thread.x = %d\n", threadIdx.x);
}

int main()
{
    printing<<<3, 2>>>(); /* <<<blk_per_grid, th_per_blk>>> */
    hipDeviceSynchronize();
}