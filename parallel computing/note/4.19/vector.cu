#include <stdio.h>
#include <stdlib.h>
#include <hip/hip_runtime.h>
#include <hip/hip_runtime_api.h>

#define SIZE 1000
#define BLKS 4
#define THREADSPBLKS 256

__global__
void vecinitKernel(int * device)
{
  int i = threadIdx.x + (blockDim.x * blockIdx.x);
  
  
  
  device[i] = i;
  
}

void vecAdd()
{
  
  int inhost[100];
  int j;
  
  int * device;
  
  for( j = 0; j < 100; j++)
    inhost[j] = 0;
  
  for( j = 0; j < 100; j++)
    printf("inhost[%d] = %d\n", j, inhost[j]);
  
  printf("---------\n");
  
  hipMalloc(&device, 100*sizeof(int));

  // kernel invocation
  vecinitKernel<<<10,10>>>(device);

  //transfer C_d from device to host
  hipMemcpy(inhost, device, 400, hipMemcpyDeviceToHost);
  
  hipFree(device);
  
  for( j = 0; j < 100; j++)
    printf("inhost[%d] = %d\n", j, inhost[j]);

}


int main()
{
   vecAdd();
   
  
    return 0;
}
